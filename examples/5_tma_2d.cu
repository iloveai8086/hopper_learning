#include "hip/hip_runtime.h"
// This code uses TMA's 2d load to load a matrix's tile to
// shared memory and then change the value in the
// shared memory and uses TMA's store to store the
// tile back to global memory. We print the result matrix to prove the
// changes are done

// note very carefully the order of the m and k coordinate in the api calls
// and note the alignment requirement of the coordinatess

#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <stdio.h>

#include "matrix_utilities.cuh"
#include "profile_utilities.cuh"
#include "tma.cuh"
#include "tma_tensor_map.cuh"

// Suppress warning about barrier in shared memory
#pragma nv_diag_suppress static_var_with_dynamic_init

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

constexpr size_t SMEM_HEIGHT = 16;
constexpr size_t SMEM_WIDTH = 8;
constexpr size_t M = 64; // Number of rows of matrix
constexpr size_t K = 32; // Number of columns of matrix
constexpr size_t gmem_len = M * K;

constexpr int m = 16; // subtile rows
constexpr int k = 8;  // subtile columns

static constexpr int buf_len = k * m;

__global__ void test(const __grid_constant__ CUtensorMap tensor_map, int x,
					 int y) {
	__shared__ alignas(128) int smem_buffer[buf_len];
	__shared__ barrier bar;

	if (threadIdx.x == 0) {
		init(&bar, blockDim.x);
	}
	__syncthreads();

	// Load data:
	uint64_t token;
	if (threadIdx.x == 0) {
		// just to demonstrate using prefetch, completely unnecessary here
		copy_async_2d_prefetch(&tensor_map, x, y);
		// call the loading api
		cde::cp_async_bulk_tensor_2d_global_to_shared(smem_buffer, &tensor_map,
													  x, y, bar);
		token = cuda::device::barrier_arrive_tx(bar, 1, sizeof(smem_buffer));
	} else {
		token = bar.arrive();
	}

	bar.wait(cuda::std::move(token));

	__syncthreads();

	// Update subtile, + 1
	for (int i = threadIdx.x; i < buf_len; i += blockDim.x) {
		smem_buffer[i] += 1;
	}

	cde::fence_proxy_async_shared_cta();
	__syncthreads();

	// Write back to global memory:
	if (threadIdx.x == 0) {
		cde::cp_async_bulk_tensor_2d_shared_to_global(&tensor_map, x, y,
													  smem_buffer);
		cde::cp_async_bulk_commit_group();
		cde::cp_async_bulk_wait_group_read<0>();
	}
	__threadfence();
	__syncthreads();
}

__global__ void kernel2(const __grid_constant__ CUtensorMap tensor_map, int x, int y) {
	// bluk tensor 的拷贝操作需要 Shared Memory 首地址对齐 128 字节。
	__shared__ alignas(128) int smem_buffer[SMEM_HEIGHT][SMEM_WIDTH];
  
	// 创建 Shared Memory 的 cuda::barrier 变量 
	#pragma nv_diag_suppress static_var_with_dynamic_init
	__shared__ barrier bar;
  
	if (threadIdx.x == 0) {
	  // 初始化 barrier 
	  init(&bar, blockDim.x);
	  // 插入 fence
	  cde::fence_proxy_async_shared_cta();    
	}
	__syncthreads();
  
	barrier::arrival_token token;
	if (threadIdx.x == 0) {
	  // 发起 TMA 二维异步拷贝操作
	  cde::cp_async_bulk_tensor_2d_global_to_shared(&smem_buffer, &tensor_map, x, y, bar);
	  // 设置同步等待点，指定需要等待的拷贝完成的字节数。
	  token = cuda::device::barrier_arrive_tx(bar, 1, sizeof(smem_buffer));
	} else {
	  // Other threads just arrive.
	  token = bar.arrive();
	}
	// 等待完成拷贝
	bar.wait(std::move(token));
  
	smem_buffer[0][threadIdx.x] += threadIdx.x;
  
	// 插入 fence
	cde::fence_proxy_async_shared_cta();
	__syncthreads();
  
	if (threadIdx.x == 0) {
	  cde::cp_async_bulk_tensor_2d_shared_to_global(&tensor_map, x, y, &smem_buffer);
	  cde::cp_async_bulk_commit_group();
	  cde::cp_async_bulk_wait_group_read<0>();
	}
  
	if (threadIdx.x == 0) {
	  (&bar)->~barrier();
	}
  }

int main() {
	// fill the host matrix
	int host_tensor[gmem_len];  // gmem_len = M * K = 64 * 32 = 2048
	// 按照smem为一个tile填满tensor，所填的tensor为tile id
	fill_tilewise(host_tensor, M, K, m, k);

	print_matrix(host_tensor, M, K);

	// copy host matrix to device
	int *tensor_ptr = nullptr;
	hipMalloc(&tensor_ptr, gmem_len * sizeof(int));
	hipMemcpy(tensor_ptr, host_tensor, gmem_len * sizeof(int),
			   hipMemcpyHostToDevice);

	// create tensor map for the matrix
	CUtensorMap tensor_map = create_2d_tensor_map(M, K, m, k, tensor_ptr);

	// launch kernel, select a tile coordinate
	// x (0 16 32 48) y (0 8 16 24) must be aligned with m(16) and k(8)
	int coordinate_m = 48;  // 48 / 16 = 3
	int coordinate_k = 24;  // 24 / 8 = 3
	// test<<<1, 128>>>(tensor_map, coordinate_k, coordinate_m);
	kernel2<<<1, 128>>>(tensor_map, coordinate_k, coordinate_m);

	cuda_check_error();

	// copy device matrix to host
	int host_gmem_tensor[gmem_len];
	hipMemcpy(host_gmem_tensor, tensor_ptr, gmem_len * sizeof(int),
			   hipMemcpyDeviceToHost);

	// verify the results
	print_matrix(host_gmem_tensor, M, K);

	return 0;
}
